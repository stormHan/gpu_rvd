#include "hip/hip_runtime.h"
/*
 * /brief Implementation of computing Restricted Voronoi Diagram.
 */

#include <cuda\cuda_rvd.h>

namespace Gpu_Rvd{

	CudaRestrictedVoronoiDiagram::CudaRestrictedVoronoiDiagram(Mesh m, Points p) : 
		vertex_(m.v_ptr()),
		vertex_nb_(m.get_vertex_nb()),
		points_(p.v_ptr()),
		points_nb_(p.get_vertex_nb()),
		facets_(m.f_ptr()),
		facet_nb_(m.get_facet_nb()),
		k_(p.get_k()),
		points_nn_(p.get_indices()),
		facets_nn_(m.get_indices()),
		dimension_(p.dimension()),
		dev_vertex_(nil),
		dev_points_(nil),
		dev_facets_(nil),
		dev_points_nn_(nil),
		dev_facets_nn_(nil),
		dev_ret(nil),
		host_ret(nil)
	{
	}

	CudaRestrictedVoronoiDiagram::CudaRestrictedVoronoiDiagram(Mesh m, Points p, index_t k, const index_t* points_nn, const index_t* facets_nn) :
		vertex_(m.v_ptr()),
		vertex_nb_(m.get_vertex_nb()),
		points_(p.v_ptr()),
		points_nb_(p.get_vertex_nb()),
		facets_(m.f_ptr()),
		facet_nb_(m.get_facet_nb()),
		k_(k),
		points_nn_(points_nn),
		facets_nn_(facets_nn),
		dimension_(p.dimension()),
		dev_vertex_(nil),
		dev_points_(nil),
		dev_facets_(nil),
		dev_points_nn_(nil),
		dev_facets_nn_(nil),
		dev_ret(nil),
		host_ret(nil)
	{
	}

	CudaRestrictedVoronoiDiagram::CudaRestrictedVoronoiDiagram(
		const double* vertex, index_t vertex_nb,
		const double* points, index_t points_nb,
		const index_t* facets, index_t facets_nb,
		index_t* points_nn, index_t k_p,
		index_t* facets_nn, index_t dim
		) :
		vertex_(vertex),
		vertex_nb_(vertex_nb),
		points_(points),
		points_nb_(points_nb),
		facets_(facets),
		facet_nb_(facets_nb),
		k_(k_p),
		points_nn_(points_nn),
		facets_nn_(facets_nn),
		dimension_(dim),
		dev_vertex_(nil),
		dev_points_(nil),
		dev_facets_(nil),
		dev_points_nn_(nil),
		dev_facets_nn_(nil),
		dev_ret(nil),
		host_ret(nil)
	{
	}

	CudaRestrictedVoronoiDiagram::~CudaRestrictedVoronoiDiagram()
	{
	}

	/*
	 * \breif Manipulates the computed RVD data.
	 */
	__device__
	void action(){

	}

	/*
	 * \brief Clips the Polygon by the middle plane defined by point i and j.
	 */
	__device__
		void clip_by_plane(
		CudaPolygon& ping,
		CudaPolygon& pong,
		double3 position_i,
		double3 position_j,
		index_t j
		){

		//reset the pong
		pong.vertex_nb = 0;

		if (ping.vertex_nb == 0)
			return;

		// Compute d = n . (2m), where n is the
		// normal vector of the bisector [i, j]
		// and m the middle point of the bisector.
		double d = 0.0;
		d = dot(add(position_i, position_j), sub(position_i, position_j));

		//The predecessor of the first vertex is the last vertex
		index_t prev_k = ping.vertex_nb - 1;

		//get the position data
		CudaVertex* prev_vk = &ping.vertex[prev_k];

		double3 prev_vertex_position = { prev_vk->x, prev_vk->y, prev_vk->z };

		//then we compute prev_vertex_position "cross" n 
		//prev_l = prev_vertex_position . n
		double prev_l = dot(prev_vertex_position, sub(position_i, position_j));

		int prev_status = sgn(2.0 * prev_l - d);

		//traverse the Vertex in this Polygon
		for (index_t k = 0; k < ping.vertex_nb; ++k){

			CudaVertex* vk = &ping.vertex[k];
			double3 vertex_position = { vk->x, vk->y, vk->z };

			double l = dot(vertex_position, sub(position_i, position_j));
			int status = sgn(2.0 * l - d);

			//If status of edge extremities differ,
			//then there is an intersection.
			if (status != prev_status && (prev_status) != 0){
				// create the intersection and update the Polyon
				CudaVertex I;

				//compute the position and weight
				double denom = 2.0 * (prev_l - l);
				double lambda1, lambda2;

				// Shit happens!
				if (m_fabs(denom) < 1e-20)
				{
					lambda1 = 0.5;
					lambda2 = 0.5;
				}
				else
				{
					lambda1 = (d - 2.0 * l) / denom;
					// Note: lambda2 is also given
					// by (2.0*l2-d)/denom
					// (but 1.0 - lambda1 is a bit
					//  faster to compute...)
					lambda2 = 1.0 - lambda1;
				}

				//Set the Position of Vertex
				I.x = lambda1 * prev_vertex_position.x + lambda2 * vertex_position.x;
				I.y = lambda1 * prev_vertex_position.y + lambda2 * vertex_position.y;
				I.z = lambda1 * prev_vertex_position.z + lambda2 * vertex_position.z;

				//Set the Weight of Vertex
				I.w = (lambda1 * prev_vk->w + lambda2 * vk->w);

				if (status > 0)
				{
					I.neigh_s = (j);
				}
				else {
					I.neigh_s = (vk->neigh_s);
				}

				//add I to pong
				pong.vertex[pong.vertex_nb] = I;
				pong.vertex_nb++;
			}
			if (status > 0)
			{
				//add vertex to pong
				pong.vertex[pong.vertex_nb] = *vk;
				pong.vertex_nb++;
			}

			prev_vk = vk;
			prev_vertex_position = vertex_position;
			prev_status = status;
			prev_l = l;
			prev_k = k;
		}
	}

	/*
	 * \brief Swaps the content of ping and pong.
	 * stores the result in ping.
	 */
	__device__
	void swap_polygon(CudaPolygon& ping, CudaPolygon& pong){
		CudaPolygon t = ping;
		ping = pong;
		pong = t;
	}


	/*
	 * \brief Intersects a polygon with a points.
	 */
	__device__
	void intersection_clip_facet_SR(
		CudaPolygon& current_polygon,
		index_t i,
		const double* points,
		index_t points_nb,
		index_t* points_nn,
		index_t k
	){
		CudaPolygon polygon_buffer;
		
		//load /memory[points] 3 times.
		double3 pi = {
			points[i * 3 + 0],
			points[i * 3 + 1],
			points[i * 3 + 2]
		};

		for (index_t t = 0; t < k; ++t){

			//load /memory[points_nn] k times.
			index_t j = points[i * k + t];

			if (i != j){
				//load /memroy[points] k * 3 times.
				double3 pj = {
					points[j * 3 + 0],
					points[j * 3 + 1],
					points[j * 3 + 2]
				};

				double dij = distance2(pi, pj);
				double R2 = 0.0;

				for (index_t tt = 0; tt < current_polygon.vertex_nb; ++tt){
					double3 pk = { current_polygon.vertex[tt].x, current_polygon.vertex[tt].y, current_polygon.vertex[tt].z };
					double dik = distance2(pi, pk);
					R2 = max(R2, dik);
				}
				if (dij > 4.1 * R2){
					return;
				}
				clip_by_plane(current_polygon, polygon_buffer, pi, pj, j);
				swap_polygon(current_polygon, polygon_buffer);
			}
		}
	}
	

	__global__
	void kernel(
		double*			vertex,		index_t			vertex_nb,
		double*			points,		index_t			points_nb,
		index_t*		facets,		index_t			facets_nb,
		index_t*		points_nn,	index_t			k_p,
		index_t*		facets_nn,	index_t			dim,
		double*			retdata
		){
		index_t tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid >= facets_nb) return;

		//load \memory[facet] 3 times.
		int3 facet_index = {
			facets[tid * dim + 0],
			facets[tid * dim + 1],
			facets[tid * dim + 2]
		};
		
		//load \memory[vertex] 9 times.
		double3 v1 = {
			vertex[facet_index.x * dim + 0],
			vertex[facet_index.x * dim + 1],
			vertex[facet_index.x * dim + 2]
		};
		double3 v2 = {
			vertex[facet_index.y * dim + 0],
			vertex[facet_index.y * dim + 1],
			vertex[facet_index.y * dim + 2]
		};
		double3 v3 = {
			vertex[facet_index.z * dim + 0],
			vertex[facet_index.z * dim + 1],
			vertex[facet_index.z * dim + 2]
		};

		CudaPolygon current_polygon;
		current_polygon.vertex_nb = 3;

		current_polygon.vertex[0].x = v1.x; current_polygon.vertex[0].y = v1.y; current_polygon.vertex[0].z = v1.z; current_polygon.vertex[0].w = 1.0;
		current_polygon.vertex[1].x = v2.x; current_polygon.vertex[1].y = v2.y; current_polygon.vertex[1].z = v2.z; current_polygon.vertex[1].w = 1.0;
		current_polygon.vertex[2].x = v3.x; current_polygon.vertex[2].y = v3.y; current_polygon.vertex[2].z = v3.z; current_polygon.vertex[2].w = 1.0;

		//doesn't have the stack?
		index_t to_visit[CUDA_Stack_size];
		index_t to_visit_pos = 0;

		index_t has_visited[CUDA_Stack_size];
		index_t has_visited_nb = 0;
		bool has_visited_flag = false;

		//load \memory[facets_nn] 1 time.
		to_visit[to_visit_pos++] = facets_nn[tid];
		has_visited[has_visited_nb++] = to_visit[0];

		while (to_visit_pos){
			index_t current_seed = to_visit[to_visit_pos - 1];
			to_visit_pos--;

			intersection_clip_facet_SR(
				current_polygon,
				current_seed,
				points,
				points_nb,
				points_nn,
				k_p
				);

			//now we get the clipped polygon stored in "polygon", do something.
			action(
				);

			//Propagate to adjacent seeds
			for (index_t v = 0; v < current_polygon.vertex_nb; ++v)
			{
				CudaVertex ve = current_polygon.vertex[v];
				int ns = ve.neigh_s;
				if (ns != -1)
				{
					for (index_t ii = 0; ii < has_visited_nb; ++ii)
					{
						//if the neighbor seed has clipped the polygon
						//the flag should be set "true"
						if (has_visited[ii] == ns)
							has_visited_flag = true;
					}
					//the neighbor seed is new!
					if (!has_visited_flag)
					{
						to_visit[to_visit_pos++] = ns;
						has_visited[has_visited_nb++] = ns;
					}
					has_visited_flag = false;
				}
			}
		}
		if (tid == 0){
			retdata[0] = current_polygon.vertex_nb;
			
		}
	}

	__host__
	void CudaRestrictedVoronoiDiagram::compute_Rvd(){
		CudaStopWatcher watcher("compute_rvd");
		watcher.start();

		allocate_and_copy(GLOBAL_MEMORY);
		//might be improved dim3 type.
		int threads = 512;
		int blocks = facet_nb_ / threads + ((facet_nb_ % threads) ? 1 : 0);
		kernel << < threads, blocks >> > (
			dev_vertex_, vertex_nb_,
			dev_points_, points_nb_,
			dev_facets_, facet_nb_,
			dev_points_nn_, k_,
			dev_facets_nn_, dimension_,
			dev_ret
			);
		CheckCUDAError("kernel function");
		copy_back();
		watcher.stop();
		watcher.synchronize();
		watcher.print_elaspsed_time(std::cout);
		
		std::string out_file("..//out//retdata.txt");
		print_return_data(out_file);
		free_memory();
	}

	__host__
	void CudaRestrictedVoronoiDiagram::allocate_and_copy(DeviceMemoryMode mode){

		host_ret = (double*)malloc(sizeof(double) * points_nb_ * 4);
		switch (mode)
		{
		case GLOBAL_MEMORY:
		{
			//Allocate
			//Input data.
			hipMalloc((void**)&dev_vertex_, DOUBLE_SIZE * vertex_nb_ * dimension_);
			hipMalloc((void**)&dev_points_, DOUBLE_SIZE * points_nb_ * dimension_);
			hipMalloc((void**)&dev_facets_, sizeof(index_t) * facet_nb_ * dimension_);
			hipMalloc((void**)&dev_points_nn_, sizeof(index_t) * points_nb_ * k_);
			hipMalloc((void**)&dev_facets_nn_, sizeof(index_t) * facet_nb_ * 1);

			//Output result.
			hipMalloc((void**)&dev_ret, sizeof(double) * points_nb_ * 4);
			CheckCUDAError("Allocating device memory");

			//Copy
			hipMemcpy(dev_vertex_, vertex_, DOUBLE_SIZE * vertex_nb_ * dimension_, hipMemcpyHostToDevice);
			hipMemcpy(dev_points_, points_, DOUBLE_SIZE * points_nb_ * dimension_, hipMemcpyHostToDevice);
			hipMemcpy(dev_facets_, facets_, sizeof(index_t) * facet_nb_ * dimension_, hipMemcpyHostToDevice);
			hipMemcpy(dev_points_nn_, points_nn_, sizeof(index_t) * points_nb_ * k_, hipMemcpyHostToDevice);
			hipMemcpy(dev_facets_nn_, facets_nn_, sizeof(index_t) * facet_nb_ * 1, hipMemcpyHostToDevice);

			CheckCUDAError("Copying data from host to device");
		}
			break;
		case CONSTANT_MEMORY:
			break;
		case TEXTURE_MEMORY:
			break;
		default:
			break;
		}
	}

	__host__
	void CudaRestrictedVoronoiDiagram::free_memory(){
		hipFree(dev_vertex_);
		hipFree(dev_points_);
		hipFree(dev_facets_);
		hipFree(dev_points_nn_);
		hipFree(dev_facets_nn_);
		hipFree(dev_ret);

		if (host_ret != nil){
			free(host_ret);
			host_ret = nil;
		}
	}

	__host__
	void CudaRestrictedVoronoiDiagram::copy_back(){
		hipMemcpy(host_ret, dev_ret, sizeof(double) * points_nb_ * 4, hipMemcpyDeviceToHost);
		CheckCUDAError("copy back");
	}

	__host__
		void CudaRestrictedVoronoiDiagram::print_return_data(std::string filename) const{
		index_t line_num = 4;
		std::ofstream f;
		f.open(filename);
		for (index_t t = 0; t < points_nb_; ++t){
			f << std::setprecision(18);
			f << "point " << t << " ";
			f << host_ret[t] << " "
				<< host_ret[t + 1] << " "
				<< host_ret[t + 2] << " "
				<< host_ret[t + 3] << " " << std::endl;
		}
		f.close();
	}
}